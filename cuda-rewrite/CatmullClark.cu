#include "hip/hip_runtime.h"
#include <omp.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "Utilities.h"
#include "Mesh.h"
#include "CatmullClark.h"

#define NUM_THREADS 256
#define TID (threadIdx.x + blockIdx.x * blockDim.x)
#define CHECK_TID(count) if (TID >= count) return;
#define EACH_ELEM(num_elems) (num_elems + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS



/*******************************************************************************
 * RefineCageHalfedges -- Applies halfedge refinement rules on the cage mesh
 *
 * This routine computes the halfedges of the control cage after one subdivision
 * step and stores them in the subd.
 *
 */
__global__ void RefineCageInner(const cc_Mesh *cage, int32_t vertexCount, int32_t edgeCount, int32_t faceCount, int32_t halfedgeCount, cc_Halfedge_SemiRegular *halfedgesOut){
    CHECK_TID(halfedgeCount)
    int32_t halfedgeID = TID;
    const int32_t twinID = ccm_HalfedgeTwinID(cage, halfedgeID);
    const int32_t prevID = ccm_HalfedgePrevID(cage, halfedgeID);
    const int32_t nextID = ccm_HalfedgeNextID(cage, halfedgeID);
    const int32_t faceID = ccm_HalfedgeFaceID(cage, halfedgeID);
    const int32_t edgeID = ccm_HalfedgeEdgeID(cage, halfedgeID);
    const int32_t prevEdgeID = ccm_HalfedgeEdgeID(cage, prevID);
    const int32_t prevTwinID = ccm_HalfedgeTwinID(cage, prevID);
    const int32_t vertexID = ccm_HalfedgeVertexID(cage, halfedgeID);
    const int32_t twinNextID = twinID >= 0 ? ccm_HalfedgeNextID(cage, twinID) : -1;
    
    cc_Halfedge_SemiRegular *newHalfedges[4] = {
        &halfedgesOut[(4 * halfedgeID + 0)],
        &halfedgesOut[(4 * halfedgeID + 1)],
        &halfedgesOut[(4 * halfedgeID + 2)],
        &halfedgesOut[(4 * halfedgeID + 3)]
    };

    // twinIDs
    newHalfedges[0]->twinID = 4 * twinNextID + 3;
    newHalfedges[1]->twinID = 4 * nextID     + 2;
    newHalfedges[2]->twinID = 4 * prevID     + 1;
    newHalfedges[3]->twinID = 4 * prevTwinID + 0;

    // edgeIDs
    newHalfedges[0]->edgeID = 2 * edgeID + (halfedgeID > twinID ? 0 : 1);
    newHalfedges[1]->edgeID = 2 * edgeCount + halfedgeID;
    newHalfedges[2]->edgeID = 2 * edgeCount + prevID;
    newHalfedges[3]->edgeID = 2 * prevEdgeID + (prevID > prevTwinID ? 1 : 0);

    // vertexIDs
    newHalfedges[0]->vertexID = vertexID;
    newHalfedges[1]->vertexID = vertexCount + faceCount + edgeID;
    newHalfedges[2]->vertexID = vertexCount + faceID;
    newHalfedges[3]->vertexID = vertexCount + faceCount + prevEdgeID;
}


void ccs__RefineCageHalfedges(cc_Subd *subd)
{
    const cc_Mesh *cage = subd->cage;
    const int32_t vertexCount = ccm_VertexCount(cage);
    const int32_t edgeCount = ccm_EdgeCount(cage);
    const int32_t faceCount = ccm_FaceCount(cage);
    const int32_t halfedgeCount = ccm_HalfedgeCount(cage);
    cc_Halfedge_SemiRegular *halfedgesOut = subd->halfedges;
    int32_t intermed = (halfedgeCount + NUM_THREADS - 1) / NUM_THREADS;

    printf("halfedgeCount %d, num_blocks %d, num_threads %d \n", halfedgeCount, intermed, NUM_THREADS);
    RefineCageInner<<<EACH_ELEM(halfedgeCount)>>>(cage, vertexCount, edgeCount, faceCount, halfedgeCount, halfedgesOut);
}

__global__ void RefineInnerHalfedges(cc_Subd *subd, int32_t depth, const cc_Mesh *cage, int32_t halfedgeCount, int32_t vertexCount, int32_t edgeCount, int32_t faceCount, int32_t stride, cc_Halfedge_SemiRegular *halfedgesOut){
    CHECK_TID(halfedgeCount)
    int32_t halfedgeID = TID;
    const int32_t twinID = ccs_HalfedgeTwinID(subd, halfedgeID, depth);
    const int32_t prevID = ccm_HalfedgePrevID_Quad(halfedgeID);
    const int32_t nextID = ccm_HalfedgeNextID_Quad(halfedgeID);
    const int32_t faceID = ccm_HalfedgeFaceID_Quad(halfedgeID);
    const int32_t edgeID = ccs_HalfedgeEdgeID(subd, halfedgeID, depth);
    const int32_t vertexID = ccs_HalfedgeVertexID(subd, halfedgeID, depth);
    const int32_t prevEdgeID = ccs_HalfedgeEdgeID(subd, prevID, depth);
    const int32_t prevTwinID = ccs_HalfedgeTwinID(subd, prevID, depth);
    const int32_t twinNextID = ccm_HalfedgeNextID_Quad(twinID);
    cc_Halfedge_SemiRegular *newHalfedges[4] = {
        &halfedgesOut[(4 * halfedgeID + 0)],
        &halfedgesOut[(4 * halfedgeID + 1)],
        &halfedgesOut[(4 * halfedgeID + 2)],
        &halfedgesOut[(4 * halfedgeID + 3)]
    };

    // twinIDs
    newHalfedges[0]->twinID = 4 * twinNextID + 3;
    newHalfedges[1]->twinID = 4 * nextID     + 2;
    newHalfedges[2]->twinID = 4 * prevID     + 1;
    newHalfedges[3]->twinID = 4 * prevTwinID + 0;

    // edgeIDs
    newHalfedges[0]->edgeID = 2 * edgeID + (halfedgeID > twinID ? 0 : 1);
    newHalfedges[1]->edgeID = 2 * edgeCount + halfedgeID;
    newHalfedges[2]->edgeID = 2 * edgeCount + prevID;
    newHalfedges[3]->edgeID = 2 * prevEdgeID + (prevID > prevTwinID ? 1 : 0);

    // vertexIDs
    newHalfedges[0]->vertexID = vertexID;
    newHalfedges[1]->vertexID = vertexCount + faceCount + edgeID;
    newHalfedges[2]->vertexID = vertexCount + faceID;
    newHalfedges[3]->vertexID = vertexCount + faceCount + prevEdgeID;
}


/*******************************************************************************
 * RefineHalfedges -- Applies halfedge refinement on the subd
 *
 * This routine computes the halfedges of the next subd level.
 *
 */
static void ccs__RefineHalfedges(cc_Subd *subd, int32_t depth)
{
    const cc_Mesh *cage = subd->cage;
    const int32_t halfedgeCount = ccm_HalfedgeCountAtDepth(cage, depth);
    const int32_t vertexCount = ccm_VertexCountAtDepth_Fast(cage, depth);
    const int32_t edgeCount = ccm_EdgeCountAtDepth_Fast(cage, depth);
    const int32_t faceCount = ccm_FaceCountAtDepth_Fast(cage, depth);
    const int32_t stride = ccs_CumulativeHalfedgeCountAtDepth(cage, depth);
    cc_Halfedge_SemiRegular *halfedgesOut = &subd->halfedges[stride];
    RefineInnerHalfedges<<<EACH_ELEM(halfedgeCount)>>>(subd, depth, cage, halfedgeCount, vertexCount, edgeCount, faceCount, stride, halfedgesOut);
}


/*******************************************************************************
 * RefineHalfedges
 *
 */
void ccs_RefineHalfedges(cc_Subd *subd)
{
    printf("Code has changed to Global Call\n");
    const int32_t maxDepth = ccs_MaxDepth(subd);

    ccs__RefineCageHalfedges(subd);

    for (int32_t depth = 1; depth < maxDepth; ++depth) {
        ccs__RefineHalfedges(subd, depth);
        hipDeviceSynchronize(); // seems to not be necessary? We'll see.
    }
}

/*******************************************************************************
 * RefineVertexPoints -- Computes the result of Catmull Clark subdivision.
 *
 */
void ccs__ClearVertexPoints(cc_Subd *subd)
{
    const int32_t vertexCount = ccs_CumulativeVertexCount(subd);
    const int32_t vertexByteCount = vertexCount * sizeof(cc_VertexPoint);

    CC_MEMSET(subd->vertexPoints, 0, vertexByteCount);
}

void ccs__CageFacePoints_Scatter(cc_Subd *subd)
{
    const cc_Mesh *cage = subd->cage;
    const int32_t vertexCount = ccm_VertexCount(cage);
    const int32_t halfedgeCount = ccm_HalfedgeCount(cage);
    cc_VertexPoint *newFacePoints = &subd->vertexPoints[vertexCount];

CC_PARALLEL_FOR
    for (int32_t halfedgeID = 0; halfedgeID < halfedgeCount; ++halfedgeID) {
        const cc_VertexPoint vertexPoint = ccm_HalfedgeVertexPoint(cage, halfedgeID);
        const int32_t faceID = ccm_HalfedgeFaceID(cage, halfedgeID);
        double faceVertexCount = 1.0f;
        double *newFacePoint = newFacePoints[faceID].array;

        for (int32_t halfedgeIt = ccm_HalfedgeNextID(cage, halfedgeID);
                     halfedgeIt != halfedgeID;
                     halfedgeIt = ccm_HalfedgeNextID(cage, halfedgeIt)) {
            ++faceVertexCount;
        }

        for (int32_t i = 0; i < 3; ++i) {
CC_ATOMIC
            newFacePoint[i]+= vertexPoint.array[i] / (double)faceVertexCount;
        }
    }
CC_BARRIER
}

void ccs__CreasedCageEdgePoints_Scatter(cc_Subd *subd)
{
    const cc_Mesh *cage = subd->cage;
    const int32_t faceCount = ccm_FaceCount(cage);
    const int32_t vertexCount = ccm_VertexCount(cage);
    const int32_t halfedgeCount = ccm_HalfedgeCount(cage);
    const cc_VertexPoint *newFacePoints = &subd->vertexPoints[vertexCount];
    cc_VertexPoint *newEdgePoints = &subd->vertexPoints[vertexCount + faceCount];

CC_PARALLEL_FOR
    for (int32_t halfedgeID = 0; halfedgeID < halfedgeCount; ++halfedgeID) {
        const int32_t faceID = ccm_HalfedgeFaceID(cage, halfedgeID);
        const int32_t edgeID = ccm_HalfedgeEdgeID(cage, halfedgeID);
        const int32_t twinID = ccm_HalfedgeTwinID(cage, halfedgeID);
        const int32_t nextID = ccm_HalfedgeNextID(cage, halfedgeID);
        const double sharp = ccm_CreaseSharpness(cage, edgeID);
        const double edgeWeight = cc__Satf(sharp);
        const cc_VertexPoint newFacePoint = newFacePoints[faceID];
        const cc_VertexPoint oldEdgePoints[2] = {
            ccm_HalfedgeVertexPoint(cage, halfedgeID),
            ccm_HalfedgeVertexPoint(cage,     nextID)
        };
        cc_VertexPoint smoothPoint = {0.0f, 0.0f, 0.0f};
        cc_VertexPoint sharpPoint = {0.0f, 0.0f, 0.0f};
        double tmp[3], atomicWeight[3];

        // sharp point
        cc__Lerp3f(tmp, oldEdgePoints[0].array, oldEdgePoints[1].array, 0.5f);
        cc__Mul3f(sharpPoint.array, tmp, twinID < 0 ? 1.0f : 0.5f);

        // smooth point
        cc__Lerp3f(tmp, oldEdgePoints[0].array, newFacePoint.array, 0.5f);
        cc__Mul3f(smoothPoint.array, tmp, 0.5f);

        // atomic weight
        cc__Lerp3f(atomicWeight,
                   smoothPoint.array,
                   sharpPoint.array,
                   edgeWeight);

        for (int32_t i = 0; i < 3; ++i) {
CC_ATOMIC
            newEdgePoints[edgeID].array[i]+= atomicWeight[i];
        }
    }
CC_BARRIER
}


void ccs__CreasedCageVertexPoints_Scatter(cc_Subd *subd)
{
    const cc_Mesh *cage = subd->cage;
    const int32_t faceCount = ccm_FaceCount(cage);
    const int32_t vertexCount = ccm_VertexCount(cage);
    const int32_t halfedgeCount = ccm_HalfedgeCount(cage);
    const cc_VertexPoint *oldVertexPoints = cage->vertexPoints;
    const cc_VertexPoint *newFacePoints = &subd->vertexPoints[vertexCount];
    const cc_VertexPoint *newEdgePoints = &subd->vertexPoints[vertexCount + faceCount];
    cc_VertexPoint *newVertexPoints = subd->vertexPoints;

CC_PARALLEL_FOR
    for (int32_t halfedgeID = 0; halfedgeID < halfedgeCount; ++halfedgeID) {
        const int32_t vertexID = ccm_HalfedgeVertexID(cage, halfedgeID);
        const int32_t edgeID = ccm_HalfedgeEdgeID(cage, halfedgeID);
        const int32_t faceID = ccm_HalfedgeFaceID(cage, halfedgeID);
        const int32_t prevID = ccm_HalfedgePrevID(cage, halfedgeID);
        const int32_t prevEdgeID = ccm_HalfedgeEdgeID(cage, prevID);
        const double thisS = ccm_HalfedgeSharpness(cage, halfedgeID);
        const double prevS = ccm_HalfedgeSharpness(cage,     prevID);
        const double creaseWeight = cc__Signf(thisS);
        const double prevCreaseWeight = cc__Signf(prevS);
        const cc_VertexPoint newPrevEdgePoint = newEdgePoints[prevEdgeID];
        const cc_VertexPoint newEdgePoint = newEdgePoints[edgeID];
        const cc_VertexPoint newFacePoint = newFacePoints[faceID];
        const cc_VertexPoint oldPoint = oldVertexPoints[vertexID];
        cc_VertexPoint cornerPoint = {0.0f, 0.0f, 0.0f};
        cc_VertexPoint smoothPoint = {0.0f, 0.0f, 0.0f};
        cc_VertexPoint creasePoint = {0.0f, 0.0f, 0.0f};
        cc_VertexPoint atomicWeight = {0.0f, 0.0f, 0.0f};
        double avgS = prevS;
        double creaseCount = prevCreaseWeight;
        double valence = 1.0f;
        int32_t forwardIterator, backwardIterator;
        double tmp1[3], tmp2[3];

        for (forwardIterator = ccm_HalfedgeTwinID(cage, prevID);
             forwardIterator >= 0 && forwardIterator != halfedgeID;
             forwardIterator = ccm_HalfedgeTwinID(cage, forwardIterator)) {
            const int32_t prevID = ccm_HalfedgePrevID(cage, forwardIterator);
            const double prevS = ccm_HalfedgeSharpness(cage, prevID);
            const double prevCreaseWeight = cc__Signf(prevS);

            // valence computation
            ++valence;

            // crease computation
            avgS+= prevS;
            creaseCount+= prevCreaseWeight;

            // next vertex halfedge
            forwardIterator = prevID;
        }

        for (backwardIterator = ccm_HalfedgeTwinID(cage, halfedgeID);
             forwardIterator < 0 && backwardIterator >= 0 && backwardIterator != halfedgeID;
             backwardIterator = ccm_HalfedgeTwinID(cage, backwardIterator)) {
            const int32_t nextID = ccm_HalfedgeNextID(cage, backwardIterator);
            const double nextS = ccm_HalfedgeSharpness(cage, nextID);
            const double nextCreaseWeight = cc__Signf(nextS);

            // valence computation
            ++valence;

            // crease computation
            avgS+= nextS;
            creaseCount+= nextCreaseWeight;

            // next vertex halfedge
            backwardIterator = nextID;
        }

        // corner point
        cc__Mul3f(cornerPoint.array, oldPoint.array, 1.0f / valence);

        // crease computation: V / 4
        cc__Mul3f(tmp1, oldPoint.array, 0.25f * creaseWeight);
        cc__Mul3f(tmp2, newEdgePoint.array, 0.25f * creaseWeight);
        cc__Add3f(creasePoint.array, tmp1, tmp2);

        // smooth computation: (4E - F + (n - 3) V) / N
        cc__Mul3f(tmp1, newFacePoint.array, -1.0f);
        cc__Mul3f(tmp2, newEdgePoint.array, +4.0f);
        cc__Add3f(smoothPoint.array, tmp1, tmp2);
        cc__Mul3f(tmp1, oldPoint.array, valence - 3.0f);
        cc__Add3f(smoothPoint.array, smoothPoint.array, tmp1);
        cc__Mul3f(smoothPoint.array,
                  smoothPoint.array,
                  1.0f / (valence * valence));

        // boundary corrections
        if (forwardIterator < 0) {
            creaseCount+= creaseWeight;
            ++valence;

            cc__Mul3f(tmp1, oldPoint.array, 0.25f * prevCreaseWeight);
            cc__Mul3f(tmp2, newPrevEdgePoint.array, 0.25f * prevCreaseWeight);
            cc__Add3f(tmp1, tmp1, tmp2);
            cc__Add3f(creasePoint.array, creasePoint.array, tmp1);
        }

        // atomicWeight (TODO: make branchless ?)
        if (creaseCount <= 1.0f) {
            atomicWeight = smoothPoint;
        } else if (creaseCount >= 3.0f || valence == 2.0f) {
            atomicWeight = cornerPoint;
        } else {
            cc__Lerp3f(atomicWeight.array,
                       cornerPoint.array,
                       creasePoint.array,
                       cc__Satf(avgS * 0.5f));
        }
        for (int32_t i = 0; i < 3; ++i) {
CC_ATOMIC
            newVertexPoints[vertexID].array[i]+= atomicWeight.array[i];
        }
    }
CC_BARRIER
}


void ccs__FacePoints_Scatter(cc_Subd *subd, int32_t depth)
{
    const cc_Mesh *cage = subd->cage;
    const int32_t halfedgeCount = ccm_HalfedgeCountAtDepth(cage, depth);
    const int32_t vertexCount = ccm_VertexCountAtDepth_Fast(cage, depth);
    const int32_t stride = ccs_CumulativeVertexCountAtDepth(cage, depth);
    cc_VertexPoint *newFacePoints = &subd->vertexPoints[stride + vertexCount];

CC_PARALLEL_FOR
    for (int32_t halfedgeID = 0; halfedgeID < halfedgeCount; ++halfedgeID) {
        const cc_VertexPoint vertexPoint = ccs_HalfedgeVertexPoint(subd, halfedgeID, depth);
        const int32_t faceID = ccs_HalfedgeFaceID(subd, halfedgeID, depth);
        double *newFacePoint = newFacePoints[faceID].array;

        for (int32_t i = 0; i < 3; ++i) {
    CC_ATOMIC
            newFacePoint[i]+= vertexPoint.array[i] / (double)4.0f;
        }
    }
CC_BARRIER
}

void ccs__CreasedEdgePoints_Scatter(cc_Subd *subd, int32_t depth)
{
    const cc_Mesh *cage = subd->cage;
    const int32_t vertexCount = ccm_VertexCountAtDepth_Fast(cage, depth);
    const int32_t faceCount = ccm_FaceCountAtDepth_Fast(cage, depth);
    const int32_t halfedgeCount = ccm_HalfedgeCountAtDepth(cage, depth);
    const int32_t stride = ccs_CumulativeVertexCountAtDepth(cage, depth);
    const cc_VertexPoint *newFacePoints = &subd->vertexPoints[stride + vertexCount];
    cc_VertexPoint *newEdgePoints = &subd->vertexPoints[stride + vertexCount + faceCount];

CC_PARALLEL_FOR
    for (int32_t halfedgeID = 0; halfedgeID < halfedgeCount; ++halfedgeID) {
        const int32_t twinID = ccs_HalfedgeTwinID(subd, halfedgeID, depth);
        const int32_t edgeID = ccs_HalfedgeEdgeID(subd, halfedgeID, depth);
        const int32_t faceID = ccs_HalfedgeFaceID(subd, halfedgeID, depth);
        const int32_t nextID = ccs_HalfedgeNextID(subd, halfedgeID, depth);
        const double sharp = ccs_CreaseSharpness(subd, edgeID, depth);
        const double edgeWeight = cc__Satf(sharp);
        const cc_VertexPoint newFacePoint = newFacePoints[faceID];
        const cc_VertexPoint oldEdgePoints[2] = {
            ccs_HalfedgeVertexPoint(subd, halfedgeID, depth),
            ccs_HalfedgeVertexPoint(subd,     nextID, depth)
        };
        cc_VertexPoint smoothPoint = {0.0f, 0.0f, 0.0f};
        cc_VertexPoint sharpPoint = {0.0f, 0.0f, 0.0f};
        double tmp[3], atomicWeight[3];

        // sharp point
        cc__Lerp3f(tmp, oldEdgePoints[0].array, oldEdgePoints[1].array, 0.5f);
        cc__Mul3f(sharpPoint.array, tmp, twinID < 0 ? 1.0f : 0.5f);

        // smooth point
        cc__Lerp3f(tmp, oldEdgePoints[0].array, newFacePoint.array, 0.5f);
        cc__Mul3f(smoothPoint.array, tmp, 0.5f);

        // atomic weight
        cc__Lerp3f(atomicWeight,
                   smoothPoint.array,
                   sharpPoint.array,
                   edgeWeight);

        for (int32_t i = 0; i < 3; ++i) {
CC_ATOMIC
            newEdgePoints[edgeID].array[i]+= atomicWeight[i];
        }
    }
CC_BARRIER
}



void ccs__CreasedVertexPoints_Scatter(cc_Subd *subd, int32_t depth)
{
    const cc_Mesh *cage = subd->cage;
    const int32_t halfedgeCount = ccm_HalfedgeCountAtDepth(cage, depth);
    const int32_t vertexCount = ccm_VertexCountAtDepth_Fast(cage, depth);
    const int32_t faceCount = ccm_FaceCountAtDepth_Fast(cage, depth);
    const int32_t stride = ccs_CumulativeVertexCountAtDepth(cage, depth);
    const cc_VertexPoint *newFacePoints = &subd->vertexPoints[stride + vertexCount];
    const cc_VertexPoint *newEdgePoints = &subd->vertexPoints[stride + vertexCount + faceCount];
    cc_VertexPoint *newVertexPoints = &subd->vertexPoints[stride];

CC_PARALLEL_FOR
    for (int32_t halfedgeID = 0; halfedgeID < halfedgeCount; ++halfedgeID) {
        const int32_t vertexID = ccs_HalfedgeVertexID(subd, halfedgeID, depth);
        const int32_t edgeID = ccs_HalfedgeEdgeID(subd, halfedgeID, depth);
        const int32_t faceID = ccs_HalfedgeFaceID(subd, halfedgeID, depth);
        const int32_t prevID = ccs_HalfedgePrevID(subd, halfedgeID, depth);
        const int32_t prevEdgeID = ccs_HalfedgeEdgeID(subd, prevID, depth);
        const double thisS = ccs_HalfedgeSharpness(subd, halfedgeID, depth);
        const double prevS = ccs_HalfedgeSharpness(subd,     prevID, depth);
        const double creaseWeight = cc__Signf(thisS);
        const double prevCreaseWeight = cc__Signf(prevS);
        const cc_VertexPoint newPrevEdgePoint = newEdgePoints[prevEdgeID];
        const cc_VertexPoint newEdgePoint = newEdgePoints[edgeID];
        const cc_VertexPoint newFacePoint = newFacePoints[faceID];
        const cc_VertexPoint oldPoint = ccs_VertexPoint(subd, vertexID, depth);
        cc_VertexPoint cornerPoint = {0.0f, 0.0f, 0.0f};
        cc_VertexPoint smoothPoint = {0.0f, 0.0f, 0.0f};
        cc_VertexPoint creasePoint = {0.0f, 0.0f, 0.0f};
        cc_VertexPoint atomicWeight = {0.0f, 0.0f, 0.0f};
        double avgS = prevS;
        double creaseCount = prevCreaseWeight;
        double valence = 1.0f;
        int32_t forwardIterator, backwardIterator;
        double tmp1[3], tmp2[3];

        for (forwardIterator = ccs_HalfedgeTwinID(subd, prevID, depth);
             forwardIterator >= 0 && forwardIterator != halfedgeID;
             forwardIterator = ccs_HalfedgeTwinID(subd, forwardIterator, depth)) {
            
            const int32_t prevID = ccs_HalfedgePrevID(subd, forwardIterator, depth);
            const double prevS = ccs_HalfedgeSharpness(subd, prevID, depth);
            const double prevCreaseWeight = cc__Signf(prevS);

            // valence computation
            ++valence;

            // crease computation
            avgS+= prevS;
            creaseCount+= prevCreaseWeight;

            // next vertex halfedge
            forwardIterator = prevID;
        }

        for (backwardIterator = ccs_HalfedgeTwinID(subd, halfedgeID, depth);
             forwardIterator < 0 && backwardIterator >= 0 && backwardIterator != halfedgeID;
             backwardIterator = ccs_HalfedgeTwinID(subd, backwardIterator, depth)) {
            const int32_t nextID = ccs_HalfedgeNextID(subd, backwardIterator, depth);
            const double nextS = ccs_HalfedgeSharpness(subd, nextID, depth);
            const double nextCreaseWeight = cc__Signf(nextS);

            // valence computation
            ++valence;

            // crease computation
            avgS+= nextS;
            creaseCount+= nextCreaseWeight;

            // next vertex halfedge
            backwardIterator = nextID;
        }

        // corner point
        cc__Mul3f(cornerPoint.array, oldPoint.array, 1.0f / valence);

        // crease computation: V / 4
        cc__Mul3f(tmp1, oldPoint.array, 0.25f * creaseWeight);
        cc__Mul3f(tmp2, newEdgePoint.array, 0.25f * creaseWeight);
        cc__Add3f(creasePoint.array, tmp1, tmp2);

        // smooth computation: (4E - F + (n - 3) V) / N
        cc__Mul3f(tmp1, newFacePoint.array, -1.0f);
        cc__Mul3f(tmp2, newEdgePoint.array, +4.0f);
        cc__Add3f(smoothPoint.array, tmp1, tmp2);
        cc__Mul3f(tmp1, oldPoint.array, valence - 3.0f);
        cc__Add3f(smoothPoint.array, smoothPoint.array, tmp1);
        cc__Mul3f(smoothPoint.array,
                  smoothPoint.array,
                  1.0f / (valence * valence));

        // boundary corrections
        if (forwardIterator < 0) {
            creaseCount+= creaseWeight;
            ++valence;

            cc__Mul3f(tmp1, oldPoint.array, 0.25f * prevCreaseWeight);
            cc__Mul3f(tmp2, newPrevEdgePoint.array, 0.25f * prevCreaseWeight);
            cc__Add3f(tmp1, tmp1, tmp2);
            cc__Add3f(creasePoint.array, creasePoint.array, tmp1);
        }

        // atomicWeight (TODO: make branchless ?)
        if (creaseCount >= 3.0f || valence == 2.0f) {
            atomicWeight = cornerPoint;
        } else if (creaseCount <= 1.0f) {
            atomicWeight = smoothPoint;
        } else {
            cc__Lerp3f(atomicWeight.array,
                       cornerPoint.array,
                       creasePoint.array,
                       cc__Satf(avgS * 0.5f));
        }

        for (int32_t i = 0; i < 3; ++i) {
CC_ATOMIC
            newVertexPoints[vertexID].array[i]+= atomicWeight.array[i];
        }
    }
CC_BARRIER
}


void ccs_RefineVertexPoints_Scatter(cc_Subd *subd)
{
    ccs__ClearVertexPoints(subd);
    ccs__CageFacePoints_Scatter(subd);
    ccs__CreasedCageEdgePoints_Scatter(subd);
    ccs__CreasedCageVertexPoints_Scatter(subd);

    for (int32_t depth = 1; depth < ccs_MaxDepth(subd); ++depth) {
        ccs__FacePoints_Scatter(subd, depth);
        ccs__CreasedEdgePoints_Scatter(subd, depth);
        ccs__CreasedVertexPoints_Scatter(subd, depth);
    }
}