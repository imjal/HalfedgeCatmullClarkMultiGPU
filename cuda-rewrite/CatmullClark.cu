#include "hip/hip_runtime.h"
#include <omp.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "Utilities.h"
#include "Mesh.h"
#include "CatmullClark.h"

#define NUM_THREADS 256
#define NUM_THREADS_PER_GPU 256
#define NUM_GPUS 1
#define NUM_ELEMS_PER_GPU(num_elems) (num_elems + NUM_GPUS - 1) / NUM_GPUS
#define EACH_ELEM_GPU(num_elems) (NUM_ELEMS_PER_GPU(num_elems) + NUM_THREADS_PER_GPU - 1) / NUM_THREADS, NUM_THREADS_PER_GPU
#define NEW_TID(device, num_elems) (threadIdx.x + blockIdx.x * blockDim.x + (NUM_ELEMS_PER_GPU(num_elems) * device))
#define TID (threadIdx.x + blockIdx.x * blockDim.x)
#define CHECK_TID(count) if (TID >= count) return;
#define CHECK_NEW_TID(id,count) if (id > count) return;
#define EACH_ELEM(num_elems) (num_elems + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS
#define GET_DEVICE(var_name)\
    int var_name;\
    hipGetDevice(&var_name);

#define CHECK_ASSIGN_TID(id, num_elems)\
    GET_DEVICE(device_num);\
    int32_t id = NEW_TID(device_num, num_elems);\
    CHECK_NEW_TID(id,num_elems)

// __host__ __device__ int32_t getNewTID(int32_t num_elems){
//     int device_num;
//     hipGetDevice(&device_num);
//     return (threadIdx.x + blockIdx.x * blockDim.x + (NUM_ELEMS_PER_GPU(num_elems) * device_num));
// }

/*******************************************************************************
 * RefineCageHalfedges -- Applies halfedge refinement rules on the cage mesh
 *
 * This routine computes the halfedges of the control cage after one subdivision
 * step and stores them in the subd.
 *
 */
__global__ void RefineCageInner(const cc_Mesh *cage, int32_t vertexCount, int32_t edgeCount, int32_t faceCount, int32_t halfedgeCount, cc_Halfedge_SemiRegular *halfedgesOut){
    CHECK_ASSIGN_TID(halfedgeID, halfedgeCount)
    const int32_t twinID = ccm_HalfedgeTwinID(cage, halfedgeID);
    const int32_t prevID = ccm_HalfedgePrevID(cage, halfedgeID);
    const int32_t nextID = ccm_HalfedgeNextID(cage, halfedgeID);
    const int32_t faceID = ccm_HalfedgeFaceID(cage, halfedgeID);
    const int32_t edgeID = ccm_HalfedgeEdgeID(cage, halfedgeID);
    const int32_t prevEdgeID = ccm_HalfedgeEdgeID(cage, prevID);
    const int32_t prevTwinID = ccm_HalfedgeTwinID(cage, prevID);
    const int32_t vertexID = ccm_HalfedgeVertexID(cage, halfedgeID);
    const int32_t twinNextID = twinID >= 0 ? ccm_HalfedgeNextID(cage, twinID) : -1;
    
    cc_Halfedge_SemiRegular *newHalfedges[4] = {
        &halfedgesOut[(4 * halfedgeID + 0)],
        &halfedgesOut[(4 * halfedgeID + 1)],
        &halfedgesOut[(4 * halfedgeID + 2)],
        &halfedgesOut[(4 * halfedgeID + 3)]
    };

    // twinIDs
    newHalfedges[0]->twinID = 4 * twinNextID + 3;
    newHalfedges[1]->twinID = 4 * nextID     + 2;
    newHalfedges[2]->twinID = 4 * prevID     + 1;
    newHalfedges[3]->twinID = 4 * prevTwinID + 0;

    // edgeIDs
    newHalfedges[0]->edgeID = 2 * edgeID + (halfedgeID > twinID ? 0 : 1);
    newHalfedges[1]->edgeID = 2 * edgeCount + halfedgeID;
    newHalfedges[2]->edgeID = 2 * edgeCount + prevID;
    newHalfedges[3]->edgeID = 2 * prevEdgeID + (prevID > prevTwinID ? 1 : 0);

    // vertexIDs
    newHalfedges[0]->vertexID = vertexID;
    newHalfedges[1]->vertexID = vertexCount + faceCount + edgeID;
    newHalfedges[2]->vertexID = vertexCount + faceID;
    newHalfedges[3]->vertexID = vertexCount + faceCount + prevEdgeID;
}


void ccs__RefineCageHalfedges(cc_Subd *subd)
{
    const cc_Mesh *cage = subd->cage;
    const int32_t vertexCount = ccm_VertexCount(cage);
    const int32_t edgeCount = ccm_EdgeCount(cage);
    const int32_t faceCount = ccm_FaceCount(cage);
    const int32_t halfedgeCount = ccm_HalfedgeCount(cage);
    cc_Halfedge_SemiRegular *halfedgesOut = subd->halfedges;

    #pragma omp parallel for
    for(int i = 0; i < NUM_GPUS; i++){
        hipSetDevice(i);
        RefineCageInner<<<EACH_ELEM_GPU(halfedgeCount)>>>(cage, vertexCount, edgeCount, faceCount, halfedgeCount, halfedgesOut);
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
    
}

__global__ void RefineInnerHalfedges(cc_Subd *subd, int32_t depth, const cc_Mesh *cage, int32_t halfedgeCount, int32_t vertexCount, int32_t edgeCount, int32_t faceCount, int32_t stride, cc_Halfedge_SemiRegular *halfedgesOut){

    CHECK_ASSIGN_TID(halfedgeID, halfedgeCount)
    const int32_t twinID = ccs_HalfedgeTwinID(subd, halfedgeID, depth);
    const int32_t prevID = ccm_HalfedgePrevID_Quad(halfedgeID);
    const int32_t nextID = ccm_HalfedgeNextID_Quad(halfedgeID);
    const int32_t faceID = ccm_HalfedgeFaceID_Quad(halfedgeID);
    const int32_t edgeID = ccs_HalfedgeEdgeID(subd, halfedgeID, depth);
    const int32_t vertexID = ccs_HalfedgeVertexID(subd, halfedgeID, depth);
    const int32_t prevEdgeID = ccs_HalfedgeEdgeID(subd, prevID, depth);
    const int32_t prevTwinID = ccs_HalfedgeTwinID(subd, prevID, depth);
    const int32_t twinNextID = ccm_HalfedgeNextID_Quad(twinID);
    cc_Halfedge_SemiRegular *newHalfedges[4] = {
        &halfedgesOut[(4 * halfedgeID + 0)],
        &halfedgesOut[(4 * halfedgeID + 1)],
        &halfedgesOut[(4 * halfedgeID + 2)],
        &halfedgesOut[(4 * halfedgeID + 3)]
    };

    // twinIDs
    newHalfedges[0]->twinID = 4 * twinNextID + 3;
    newHalfedges[1]->twinID = 4 * nextID     + 2;
    newHalfedges[2]->twinID = 4 * prevID     + 1;
    newHalfedges[3]->twinID = 4 * prevTwinID + 0;

    // edgeIDs
    newHalfedges[0]->edgeID = 2 * edgeID + (halfedgeID > twinID ? 0 : 1);
    newHalfedges[1]->edgeID = 2 * edgeCount + halfedgeID;
    newHalfedges[2]->edgeID = 2 * edgeCount + prevID;
    newHalfedges[3]->edgeID = 2 * prevEdgeID + (prevID > prevTwinID ? 1 : 0);

    // vertexIDs
    newHalfedges[0]->vertexID = vertexID;
    newHalfedges[1]->vertexID = vertexCount + faceCount + edgeID;
    newHalfedges[2]->vertexID = vertexCount + faceID;
    newHalfedges[3]->vertexID = vertexCount + faceCount + prevEdgeID;
}


/*******************************************************************************
 * RefineHalfedges -- Applies halfedge refinement on the subd
 *
 * This routine computes the halfedges of the next subd level.
 *
 */
static void ccs__RefineHalfedges(cc_Subd *subd, int32_t depth)
{
    const cc_Mesh *cage = subd->cage;
    const int32_t halfedgeCount = ccm_HalfedgeCountAtDepth(cage, depth);
    const int32_t vertexCount = ccm_VertexCountAtDepth_Fast(cage, depth);
    const int32_t edgeCount = ccm_EdgeCountAtDepth_Fast(cage, depth);
    const int32_t faceCount = ccm_FaceCountAtDepth_Fast(cage, depth);
    const int32_t stride = ccs_CumulativeHalfedgeCountAtDepth(cage, depth);
    cc_Halfedge_SemiRegular *halfedgesOut = &subd->halfedges[stride];

    #pragma omp parallel for
    for(int i = 0; i < NUM_GPUS; i++){
        hipSetDevice(i);
        printf("It is using cuda!!\n");
        RefineInnerHalfedges<<<EACH_ELEM_GPU(halfedgeCount)>>>(subd, depth, cage, halfedgeCount, vertexCount, edgeCount, faceCount, stride, halfedgesOut);
    }
   
}


/*******************************************************************************
 * RefineHalfedges
 *
 */
void ccs_RefineHalfedges(cc_Subd *subd)
{
    const int32_t maxDepth = ccs_MaxDepth(subd);

    ccs__RefineCageHalfedges(subd);
    hipDeviceSynchronize();

    for (int32_t depth = 1; depth < maxDepth; ++depth) {
        ccs__RefineHalfedges(subd, depth);
        hipDeviceSynchronize(); // seems to not be necessary? We'll see.
    }
}

/*******************************************************************************
 * RefineVertexPoints -- Computes the result of Catmull Clark subdivision.
 *
 */
void ccs__ClearVertexPoints(cc_Subd *subd)
{
    const int32_t vertexCount = ccs_CumulativeVertexCount(subd);
    const int32_t vertexByteCount = vertexCount * sizeof(cc_VertexPoint);

    CC_MEMSET(subd->vertexPoints, 0, vertexByteCount);
}


__global__ void ccs__CageFacePoints_Scatter_Inner(const cc_Mesh *cage, int32_t vertexCount, int32_t halfedgeCount, cc_VertexPoint *newFacePoints)
{
    CHECK_TID(halfedgeCount)
    int32_t halfedgeID = TID;
    // CHECK_ASSIGN_TID(halfedgeID, halfedgeCount)
    
    const cc_VertexPoint vertexPoint = ccm_HalfedgeVertexPoint(cage, halfedgeID);
    const int32_t faceID = ccm_HalfedgeFaceID(cage, halfedgeID);
    double faceVertexCount = 1.0f;
    double *newFacePoint = newFacePoints[faceID].array;

    for (int32_t halfedgeIt = ccm_HalfedgeNextID(cage, halfedgeID);
                    halfedgeIt != halfedgeID;
                    halfedgeIt = ccm_HalfedgeNextID(cage, halfedgeIt)) {
        ++faceVertexCount;
    }

    for (int32_t i = 0; i < 3; ++i) {
// CC_ATOMIC
        // newFacePoint[i]+= vertexPoint.array[i] / (double)faceVertexCount;
        atomicAdd(newFacePoint + i, vertexPoint.array[i] / (double)faceVertexCount);
    }
}

void ccs__CageFacePoints_Scatter(cc_Subd *subd)
{
    const cc_Mesh *cage = subd->cage;
    const int32_t vertexCount = ccm_VertexCount(cage);
    const int32_t halfedgeCount = ccm_HalfedgeCount(cage);
    cc_VertexPoint *newFacePoints = &subd->vertexPoints[vertexCount];

    // #pragma omp parallel for
    // for(int i = 0; i < NUM_GPUS; i++){
        // hipSetDevice(i);
        ccs__CageFacePoints_Scatter_Inner<<<EACH_ELEM(halfedgeCount)>>>(cage, vertexCount, halfedgeCount, newFacePoints);
    // }
}

__global__ void ccs__CreasedCageEdgePoints_Scatter_Inner(const cc_Mesh *cage, int32_t faceCount, int32_t vertexCount, int32_t halfedgeCount, const cc_VertexPoint *newFacePoints, cc_VertexPoint *newEdgePoints)
{
    CHECK_TID(halfedgeCount)
    // CHECK_ASSIGN_TID(halfedgeID, halfedgeCount)
    int32_t halfedgeID = TID;
    const int32_t faceID = ccm_HalfedgeFaceID(cage, halfedgeID);
    const int32_t edgeID = ccm_HalfedgeEdgeID(cage, halfedgeID);
    const int32_t twinID = ccm_HalfedgeTwinID(cage, halfedgeID);
    const int32_t nextID = ccm_HalfedgeNextID(cage, halfedgeID);
    const double sharp = ccm_CreaseSharpness(cage, edgeID);
    const double edgeWeight = cc__Satf(sharp);
    const cc_VertexPoint newFacePoint = newFacePoints[faceID];
    const cc_VertexPoint oldEdgePoints[2] = {
        ccm_HalfedgeVertexPoint(cage, halfedgeID),
        ccm_HalfedgeVertexPoint(cage,     nextID)
    };
    cc_VertexPoint smoothPoint = {0.0f, 0.0f, 0.0f};
    cc_VertexPoint sharpPoint = {0.0f, 0.0f, 0.0f};
    double tmp[3], atomicWeight[3];

    // sharp point
    cc__Lerp3f(tmp, oldEdgePoints[0].array, oldEdgePoints[1].array, 0.5f);
    cc__Mul3f(sharpPoint.array, tmp, twinID < 0 ? 1.0f : 0.5f);

    // smooth point
    cc__Lerp3f(tmp, oldEdgePoints[0].array, newFacePoint.array, 0.5f);
    cc__Mul3f(smoothPoint.array, tmp, 0.5f);

    // atomic weight
    cc__Lerp3f(atomicWeight,
                smoothPoint.array,
                sharpPoint.array,
                edgeWeight);

    for (int32_t i = 0; i < 3; ++i) {
        atomicAdd(newEdgePoints[edgeID].array + i, atomicWeight[i]);
    }
}

void ccs__CreasedCageEdgePoints_Scatter(cc_Subd *subd)
{
    const cc_Mesh *cage = subd->cage;
    const int32_t faceCount = ccm_FaceCount(cage);
    const int32_t vertexCount = ccm_VertexCount(cage);
    const int32_t halfedgeCount = ccm_HalfedgeCount(cage);
    const cc_VertexPoint *newFacePoints = &subd->vertexPoints[vertexCount];
    cc_VertexPoint *newEdgePoints = &subd->vertexPoints[vertexCount + faceCount];

    // #pragma omp for
    // for(int i = 0; i < NUM_GPUS; i++){
        // hipSetDevice(i);
        ccs__CreasedCageEdgePoints_Scatter_Inner<<<EACH_ELEM(halfedgeCount)>>>(cage, faceCount, vertexCount, halfedgeCount, newFacePoints, newEdgePoints);
    // }
}

__global__ void ccs__CreasedCageVertexPoints_Scatter_Inner(
    const cc_Mesh *cage, int32_t faceCount, const int32_t vertexCount, int32_t halfedgeCount, 
    const cc_VertexPoint *oldVertexPoints, const cc_VertexPoint *newFacePoints, 
    const cc_VertexPoint *newEdgePoints, cc_VertexPoint *newVertexPoints)
{
    CHECK_TID(halfedgeCount)
    // CHECK_ASSIGN_TID(halfedgeID, halfedgeCount)
    int32_t halfedgeID = TID;
    const int32_t vertexID = ccm_HalfedgeVertexID(cage, halfedgeID);
    const int32_t edgeID = ccm_HalfedgeEdgeID(cage, halfedgeID);
    const int32_t faceID = ccm_HalfedgeFaceID(cage, halfedgeID);
    const int32_t prevID = ccm_HalfedgePrevID(cage, halfedgeID);
    const int32_t prevEdgeID = ccm_HalfedgeEdgeID(cage, prevID);
    const double thisS = ccm_HalfedgeSharpness(cage, halfedgeID);
    const double prevS = ccm_HalfedgeSharpness(cage,     prevID);
    const double creaseWeight = cc__Signf(thisS);
    const double prevCreaseWeight = cc__Signf(prevS);
    const cc_VertexPoint newPrevEdgePoint = newEdgePoints[prevEdgeID];
    const cc_VertexPoint newEdgePoint = newEdgePoints[edgeID];
    const cc_VertexPoint newFacePoint = newFacePoints[faceID];
    const cc_VertexPoint oldPoint = oldVertexPoints[vertexID];
    cc_VertexPoint cornerPoint = {0.0f, 0.0f, 0.0f};
    cc_VertexPoint smoothPoint = {0.0f, 0.0f, 0.0f};
    cc_VertexPoint creasePoint = {0.0f, 0.0f, 0.0f};
    cc_VertexPoint atomicWeight = {0.0f, 0.0f, 0.0f};
    double avgS = prevS;
    double creaseCount = prevCreaseWeight;
    double valence = 1.0f;
    int32_t forwardIterator, backwardIterator;
    double tmp1[3], tmp2[3];

    for (forwardIterator = ccm_HalfedgeTwinID(cage, prevID);
            forwardIterator >= 0 && forwardIterator != halfedgeID;
            forwardIterator = ccm_HalfedgeTwinID(cage, forwardIterator)) {
        const int32_t prevID = ccm_HalfedgePrevID(cage, forwardIterator);
        const double prevS = ccm_HalfedgeSharpness(cage, prevID);
        const double prevCreaseWeight = cc__Signf(prevS);

        // valence computation
        ++valence;

        // crease computation
        avgS+= prevS;
        creaseCount+= prevCreaseWeight;

        // next vertex halfedge
        forwardIterator = prevID;
    }

    for (backwardIterator = ccm_HalfedgeTwinID(cage, halfedgeID);
            forwardIterator < 0 && backwardIterator >= 0 && backwardIterator != halfedgeID;
            backwardIterator = ccm_HalfedgeTwinID(cage, backwardIterator)) {
        const int32_t nextID = ccm_HalfedgeNextID(cage, backwardIterator);
        const double nextS = ccm_HalfedgeSharpness(cage, nextID);
        const double nextCreaseWeight = cc__Signf(nextS);

        // valence computation
        ++valence;

        // crease computation
        avgS+= nextS;
        creaseCount+= nextCreaseWeight;

        // next vertex halfedge
        backwardIterator = nextID;
    }

    // corner point
    cc__Mul3f(cornerPoint.array, oldPoint.array, 1.0f / valence);

    // crease computation: V / 4
    cc__Mul3f(tmp1, oldPoint.array, 0.25f * creaseWeight);
    cc__Mul3f(tmp2, newEdgePoint.array, 0.25f * creaseWeight);
    cc__Add3f(creasePoint.array, tmp1, tmp2);

    // smooth computation: (4E - F + (n - 3) V) / N
    cc__Mul3f(tmp1, newFacePoint.array, -1.0f);
    cc__Mul3f(tmp2, newEdgePoint.array, +4.0f);
    cc__Add3f(smoothPoint.array, tmp1, tmp2);
    cc__Mul3f(tmp1, oldPoint.array, valence - 3.0f);
    cc__Add3f(smoothPoint.array, smoothPoint.array, tmp1);
    cc__Mul3f(smoothPoint.array,
                smoothPoint.array,
                1.0f / (valence * valence));

    // boundary corrections
    if (forwardIterator < 0) {
        creaseCount+= creaseWeight;
        ++valence;

        cc__Mul3f(tmp1, oldPoint.array, 0.25f * prevCreaseWeight);
        cc__Mul3f(tmp2, newPrevEdgePoint.array, 0.25f * prevCreaseWeight);
        cc__Add3f(tmp1, tmp1, tmp2);
        cc__Add3f(creasePoint.array, creasePoint.array, tmp1);
    }

    // atomicWeight (TODO: make branchless ?)
    if (creaseCount <= 1.0f) {
        atomicWeight = smoothPoint;
    } else if (creaseCount >= 3.0f || valence == 2.0f) {
        atomicWeight = cornerPoint;
    } else {
        cc__Lerp3f(atomicWeight.array,
                    cornerPoint.array,
                    creasePoint.array,
                    cc__Satf(avgS * 0.5f));
    }
    for (int32_t i = 0; i < 3; ++i) {
        atomicAdd(newVertexPoints[vertexID].array + i, atomicWeight.array[i]);
        // newVertexPoints[vertexID].array[i]+= atomicWeight.array[i];
    }
}


void ccs__CreasedCageVertexPoints_Scatter(cc_Subd *subd)
{
    const cc_Mesh *cage = subd->cage;
    const int32_t faceCount = ccm_FaceCount(cage);
    const int32_t vertexCount = ccm_VertexCount(cage);
    const int32_t halfedgeCount = ccm_HalfedgeCount(cage);
    const cc_VertexPoint *oldVertexPoints = cage->vertexPoints;
    const cc_VertexPoint *newFacePoints = &subd->vertexPoints[vertexCount];
    const cc_VertexPoint *newEdgePoints = &subd->vertexPoints[vertexCount + faceCount];
    cc_VertexPoint *newVertexPoints = subd->vertexPoints;

    ccs__CreasedCageVertexPoints_Scatter_Inner<<<EACH_ELEM(halfedgeCount)>>>(cage, faceCount,vertexCount, halfedgeCount, oldVertexPoints,newFacePoints, newEdgePoints, newVertexPoints);
}


__global__ void ccs__FacePoints_Scatter(const cc_Subd *subd, int32_t depth, const cc_Mesh *cage, int32_t halfedgeCount, int32_t vertexCount, int32_t stride, cc_VertexPoint *newFacePoints)
{
    CHECK_TID(halfedgeCount)
    // CHECK_ASSIGN_TID(halfedgeID, halfedgeCount)
    int32_t halfedgeID = TID;
    const cc_VertexPoint vertexPoint = ccs_HalfedgeVertexPoint(subd, halfedgeID, depth);
    const int32_t faceID = ccs_HalfedgeFaceID(subd, halfedgeID, depth);
    double *newFacePoint = newFacePoints[faceID].array;

    for (int32_t i = 0; i < 3; ++i) {
        // newFacePoint[i]+= vertexPoint.array[i] / (double)4.0f;
        atomicAdd(newFacePoint + i, vertexPoint.array[i] / (double)4.0f);
    }
}


void ccs__FacePoints_Scatter(cc_Subd *subd, int32_t depth)
{
    const cc_Mesh *cage = subd->cage;
    const int32_t halfedgeCount = ccm_HalfedgeCountAtDepth(cage, depth);
    const int32_t vertexCount = ccm_VertexCountAtDepth_Fast(cage, depth);
    const int32_t stride = ccs_CumulativeVertexCountAtDepth(cage, depth);
    cc_VertexPoint *newFacePoints = &subd->vertexPoints[stride + vertexCount];

    // #pragma omp for
    // for(int i = 0; i < NUM_GPUS; i++){
        // hipSetDevice(i);
        ccs__FacePoints_Scatter<<<EACH_ELEM(halfedgeCount)>>>(subd, depth, cage, halfedgeCount, vertexCount, stride, newFacePoints);
    // }
}

__global__ void ccs__CreasedEdgePoints_Scatter(const cc_Subd *subd, int32_t depth, const cc_Mesh *cage, int32_t halfedgeCount, int32_t faceCount, int32_t vertexCount, const cc_VertexPoint *newFacePoints, cc_VertexPoint *newEdgePoints)
{
    CHECK_TID(halfedgeCount)
    // CHECK_ASSIGN_TID(halfedgeID, halfedgeCount)
    int32_t halfedgeID = TID;
    const int32_t twinID = ccs_HalfedgeTwinID(subd, halfedgeID, depth);
    const int32_t edgeID = ccs_HalfedgeEdgeID(subd, halfedgeID, depth);
    const int32_t faceID = ccs_HalfedgeFaceID(subd, halfedgeID, depth);
    const int32_t nextID = ccs_HalfedgeNextID(subd, halfedgeID, depth);
    const double sharp = ccs_CreaseSharpness(subd, edgeID, depth);
    const double edgeWeight = cc__Satf(sharp);
    const cc_VertexPoint newFacePoint = newFacePoints[faceID];
    const cc_VertexPoint oldEdgePoints[2] = {
        ccs_HalfedgeVertexPoint(subd, halfedgeID, depth),
        ccs_HalfedgeVertexPoint(subd,     nextID, depth)
    };
    cc_VertexPoint smoothPoint = {0.0f, 0.0f, 0.0f};
    cc_VertexPoint sharpPoint = {0.0f, 0.0f, 0.0f};
    double tmp[3], atomicWeight[3];

    // sharp point
    cc__Lerp3f(tmp, oldEdgePoints[0].array, oldEdgePoints[1].array, 0.5f);
    cc__Mul3f(sharpPoint.array, tmp, twinID < 0 ? 1.0f : 0.5f);

    // smooth point
    cc__Lerp3f(tmp, oldEdgePoints[0].array, newFacePoint.array, 0.5f);
    cc__Mul3f(smoothPoint.array, tmp, 0.5f);

    // atomic weight
    cc__Lerp3f(atomicWeight,
                smoothPoint.array,
                sharpPoint.array,
                edgeWeight);

    for (int32_t i = 0; i < 3; ++i) {
// CC_ATOMIC
//         newEdgePoints[edgeID].array[i]+= atomicWeight[i];
        atomicAdd(newEdgePoints[edgeID].array + i, atomicWeight[i]);
    }
}

void ccs__CreasedEdgePoints_Scatter(cc_Subd *subd, int32_t depth)
{
    const cc_Mesh *cage = subd->cage;
    const int32_t vertexCount = ccm_VertexCountAtDepth_Fast(cage, depth);
    const int32_t faceCount = ccm_FaceCountAtDepth_Fast(cage, depth);
    const int32_t halfedgeCount = ccm_HalfedgeCountAtDepth(cage, depth);
    const int32_t stride = ccs_CumulativeVertexCountAtDepth(cage, depth);
    const cc_VertexPoint *newFacePoints = &subd->vertexPoints[stride + vertexCount];
    cc_VertexPoint *newEdgePoints = &subd->vertexPoints[stride + vertexCount + faceCount];

    // #pragma omp for
    // for(int i = 0; i < NUM_GPUS; i++){
        // hipSetDevice(i);
        ccs__CreasedEdgePoints_Scatter<<<EACH_ELEM(halfedgeCount)>>>(subd, depth, cage, halfedgeCount, faceCount, vertexCount, newFacePoints, newEdgePoints);
    // }
}

__global__ void ccs__CreasedVertexPoints_Scatter(cc_Subd *subd, int32_t depth, const cc_Mesh *cage, int32_t halfedgeCount, int32_t faceCount, int32_t vertexCount, const cc_VertexPoint *newFacePoints, const cc_VertexPoint *newEdgePoints, cc_VertexPoint *newVertexPoints)
{   
    CHECK_TID(halfedgeCount)
    // CHECK_ASSIGN_TID(halfedgeID, halfedgeCount)
    int32_t halfedgeID = TID;
    const int32_t vertexID = ccs_HalfedgeVertexID(subd, halfedgeID, depth);
    const int32_t edgeID = ccs_HalfedgeEdgeID(subd, halfedgeID, depth);
    const int32_t faceID = ccs_HalfedgeFaceID(subd, halfedgeID, depth);
    const int32_t prevID = ccs_HalfedgePrevID(subd, halfedgeID, depth);
    const int32_t prevEdgeID = ccs_HalfedgeEdgeID(subd, prevID, depth);
    const double thisS = ccs_HalfedgeSharpness(subd, halfedgeID, depth);
    const double prevS = ccs_HalfedgeSharpness(subd,     prevID, depth);
    const double creaseWeight = cc__Signf(thisS);
    const double prevCreaseWeight = cc__Signf(prevS);
    const cc_VertexPoint newPrevEdgePoint = newEdgePoints[prevEdgeID];
    const cc_VertexPoint newEdgePoint = newEdgePoints[edgeID];
    const cc_VertexPoint newFacePoint = newFacePoints[faceID];
    const cc_VertexPoint oldPoint = ccs_VertexPoint(subd, vertexID, depth);
    cc_VertexPoint cornerPoint = {0.0f, 0.0f, 0.0f};
    cc_VertexPoint smoothPoint = {0.0f, 0.0f, 0.0f};
    cc_VertexPoint creasePoint = {0.0f, 0.0f, 0.0f};
    cc_VertexPoint atomicWeight = {0.0f, 0.0f, 0.0f};
    double avgS = prevS;
    double creaseCount = prevCreaseWeight;
    double valence = 1.0f;
    int32_t forwardIterator, backwardIterator;
    double tmp1[3], tmp2[3];

    for (forwardIterator = ccs_HalfedgeTwinID(subd, prevID, depth);
            forwardIterator >= 0 && forwardIterator != halfedgeID;
            forwardIterator = ccs_HalfedgeTwinID(subd, forwardIterator, depth)) {
        
        const int32_t prevID = ccs_HalfedgePrevID(subd, forwardIterator, depth);
        const double prevS = ccs_HalfedgeSharpness(subd, prevID, depth);
        const double prevCreaseWeight = cc__Signf(prevS);

        // valence computation
        ++valence;

        // crease computation
        avgS+= prevS;
        creaseCount+= prevCreaseWeight;

        // next vertex halfedge
        forwardIterator = prevID;
    }

    for (backwardIterator = ccs_HalfedgeTwinID(subd, halfedgeID, depth);
            forwardIterator < 0 && backwardIterator >= 0 && backwardIterator != halfedgeID;
            backwardIterator = ccs_HalfedgeTwinID(subd, backwardIterator, depth)) {
        const int32_t nextID = ccs_HalfedgeNextID(subd, backwardIterator, depth);
        const double nextS = ccs_HalfedgeSharpness(subd, nextID, depth);
        const double nextCreaseWeight = cc__Signf(nextS);

        // valence computation
        ++valence;

        // crease computation
        avgS+= nextS;
        creaseCount+= nextCreaseWeight;

        // next vertex halfedge
        backwardIterator = nextID;
    }

    // corner point
    cc__Mul3f(cornerPoint.array, oldPoint.array, 1.0f / valence);

    // crease computation: V / 4
    cc__Mul3f(tmp1, oldPoint.array, 0.25f * creaseWeight);
    cc__Mul3f(tmp2, newEdgePoint.array, 0.25f * creaseWeight);
    cc__Add3f(creasePoint.array, tmp1, tmp2);

    // smooth computation: (4E - F + (n - 3) V) / N
    cc__Mul3f(tmp1, newFacePoint.array, -1.0f);
    cc__Mul3f(tmp2, newEdgePoint.array, +4.0f);
    cc__Add3f(smoothPoint.array, tmp1, tmp2);
    cc__Mul3f(tmp1, oldPoint.array, valence - 3.0f);
    cc__Add3f(smoothPoint.array, smoothPoint.array, tmp1);
    cc__Mul3f(smoothPoint.array,
                smoothPoint.array,
                1.0f / (valence * valence));

    // boundary corrections
    if (forwardIterator < 0) {
        creaseCount+= creaseWeight;
        ++valence;

        cc__Mul3f(tmp1, oldPoint.array, 0.25f * prevCreaseWeight);
        cc__Mul3f(tmp2, newPrevEdgePoint.array, 0.25f * prevCreaseWeight);
        cc__Add3f(tmp1, tmp1, tmp2);
        cc__Add3f(creasePoint.array, creasePoint.array, tmp1);
    }

    // atomicWeight (TODO: make branchless ?)
    if (creaseCount >= 3.0f || valence == 2.0f) {
        atomicWeight = cornerPoint;
    } else if (creaseCount <= 1.0f) {
        atomicWeight = smoothPoint;
    } else {
        cc__Lerp3f(atomicWeight.array,
                    cornerPoint.array,
                    creasePoint.array,
                    cc__Satf(avgS * 0.5f));
    }

    for (int32_t i = 0; i < 3; ++i) {
// CC_ATOMIC
        // newVertexPoints[vertexID].array[i]+= atomicWeight.array[i];
        atomicAdd(newVertexPoints[vertexID].array + i, atomicWeight.array[i]);
    }
}


void ccs__CreasedVertexPoints_Scatter(cc_Subd *subd, int32_t depth)
{
    const cc_Mesh *cage = subd->cage;
    const int32_t halfedgeCount = ccm_HalfedgeCountAtDepth(cage, depth);
    const int32_t vertexCount = ccm_VertexCountAtDepth_Fast(cage, depth);
    const int32_t faceCount = ccm_FaceCountAtDepth_Fast(cage, depth);
    const int32_t stride = ccs_CumulativeVertexCountAtDepth(cage, depth);
    const cc_VertexPoint *newFacePoints = &subd->vertexPoints[stride + vertexCount];
    const cc_VertexPoint *newEdgePoints = &subd->vertexPoints[stride + vertexCount + faceCount];
    cc_VertexPoint *newVertexPoints = &subd->vertexPoints[stride];
    
    // #pragma omp for
    // for(int i = 0; i < NUM_GPUS; i++){
        // hipSetDevice(i);
        ccs__CreasedVertexPoints_Scatter<<<EACH_ELEM(halfedgeCount)>>>(subd, depth, cage, halfedgeCount, faceCount, vertexCount, newFacePoints, newEdgePoints, newVertexPoints);
    // }
}


void ccs_RefineVertexPoints_Scatter(cc_Subd *subd)
{
    ccs__ClearVertexPoints(subd);
    ccs__CageFacePoints_Scatter(subd);
    ccs__CreasedCageEdgePoints_Scatter(subd);
    ccs__CreasedCageVertexPoints_Scatter(subd);
    hipDeviceSynchronize();

    

    for (int32_t depth = 1; depth < ccs_MaxDepth(subd); ++depth) {
        ccs__FacePoints_Scatter(subd, depth);
        ccs__CreasedEdgePoints_Scatter(subd, depth);
        ccs__CreasedVertexPoints_Scatter(subd, depth);
        hipDeviceSynchronize();
    }
}
/*************

Start Creases Code

**************/

__global__ void ccs__RefineCageCreases_Inner(const cc_Mesh *cage, int32_t edgeCount, cc_Crease *creasesOut){
    // CHECK_TID(edgeCount)
    // int32_t edgeID = TID;
    CHECK_ASSIGN_TID(edgeID, edgeCount)

    const int32_t nextID = ccm_CreaseNextID(cage, edgeID);
    const int32_t prevID = ccm_CreasePrevID(cage, edgeID);
    const bool t1 = ccm_CreasePrevID(cage, nextID) == edgeID && nextID != edgeID;
    const bool t2 = ccm_CreaseNextID(cage, prevID) == edgeID && prevID != edgeID;
    const double thisS = 3.0f * ccm_CreaseSharpness(cage, edgeID);
    const double nextS = ccm_CreaseSharpness(cage, nextID);
    const double prevS = ccm_CreaseSharpness(cage, prevID);
    cc_Crease *newCreases[2] = {
        &creasesOut[(2 * edgeID + 0)],
        &creasesOut[(2 * edgeID + 1)]
    };

    // next rule
    newCreases[0]->nextID = 2 * edgeID + 1;
    newCreases[1]->nextID = 2 * nextID + (t1 ? 0 : 1);

    // prev rule
    newCreases[0]->prevID = 2 * prevID + (t2 ? 1 : 0);
    newCreases[1]->prevID = 2 * edgeID + 0;

    // sharpness rule
    newCreases[0]->sharpness = cc__Maxf(0.0f, (prevS + thisS) / 4.0f - 1.0f);
    newCreases[1]->sharpness = cc__Maxf(0.0f, (thisS + nextS) / 4.0f - 1.0f);
}

void ccs__RefineCageCreases(cc_Subd *subd)
{
    const cc_Mesh *cage = subd->cage;
    const int32_t edgeCount = ccm_EdgeCount(cage);
    cc_Crease *creasesOut = subd->creases;

    #pragma omp parallel for
    for(int i = 0; i < NUM_GPUS; i++){
        hipSetDevice(i);
        ccs__RefineCageCreases_Inner<<<EACH_ELEM_GPU(edgeCount)>>>(cage, edgeCount, creasesOut);
    }
}

__global__ void ccs__RefineCreases(cc_Subd *subd, int32_t depth, const cc_Mesh *cage, int32_t creaseCount, int32_t stride, cc_Crease *creasesOut)
{
    // CHECK_TID(creaseCount)
    CHECK_ASSIGN_TID(edgeID, creaseCount)
    // int32_t edgeID = TID;
    const int32_t nextID = ccs_CreaseNextID_Fast(subd, edgeID, depth);
    const int32_t prevID = ccs_CreasePrevID_Fast(subd, edgeID, depth);
    const bool t1 = ccs_CreasePrevID_Fast(subd, nextID, depth) == edgeID && nextID != edgeID;
    const bool t2 = ccs_CreaseNextID_Fast(subd, prevID, depth) == edgeID && prevID != edgeID;
    const double thisS = 3.0f * ccs_CreaseSharpness_Fast(subd, edgeID, depth);
    const double nextS = ccs_CreaseSharpness_Fast(subd, nextID, depth);
    const double prevS = ccs_CreaseSharpness_Fast(subd, prevID, depth);
    cc_Crease *newCreases[2] = {
        &creasesOut[(2 * edgeID + 0)],
        &creasesOut[(2 * edgeID + 1)]
    };

    // next rule
    newCreases[0]->nextID = 2 * edgeID + 1;
    newCreases[1]->nextID = 2 * nextID + (t1 ? 0 : 1);

    // prev rule
    newCreases[0]->prevID = 2 * prevID + (t2 ? 1 : 0);
    newCreases[1]->prevID = 2 * edgeID + 0;

    // sharpness rule
    newCreases[0]->sharpness = cc__Maxf(0.0f, (prevS + thisS) / 4.0f - 1.0f);
    newCreases[1]->sharpness = cc__Maxf(0.0f, (thisS + nextS) / 4.0f - 1.0f);
}

/*******************************************************************************
 * RefineCreases -- Applies crease subdivision on the subd
 *
 * This routine computes the topology of the next subd level.
 *
 */
void ccs__RefineCreases(cc_Subd *subd, int32_t depth)
{
    const cc_Mesh *cage = subd->cage;
    const int32_t creaseCount = ccm_CreaseCountAtDepth(cage, depth);
    const int32_t stride = ccs_CumulativeCreaseCountAtDepth(cage, depth);
    cc_Crease *creasesOut = &subd->creases[stride];
    #pragma omp parallel for
    for(int i = 0; i < NUM_GPUS; i++){
        hipSetDevice(i);
        ccs__RefineCreases<<<EACH_ELEM_GPU(creaseCount)>>>(subd, depth, cage, creaseCount, stride, creasesOut);
    }
}

void ccs_RefineCreases(cc_Subd *subd)
{
    const int32_t maxDepth = ccs_MaxDepth(subd);

    ccs__RefineCageCreases(subd);
    hipDeviceSynchronize();

    for (int32_t depth = 1; depth < maxDepth; ++depth) {
        ccs__RefineCreases(subd, depth);
        hipDeviceSynchronize();
    }
}
